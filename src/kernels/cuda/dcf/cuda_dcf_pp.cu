#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

#include "cuda_dcf_pp.h"


__global__ 
static void compute()
{
    printf("Hello DCF_PP thread %d, block %d\n", threadIdx.x, blockIdx.x);
}


__host__
void K_CUDA_DCF_PP(
    int target_x_low_ind,  int target_x_high_ind,
    int target_y_low_ind,  int target_y_high_ind,
    int target_z_low_ind,  int target_z_high_ind,
    double target_xmin,    double target_ymin,    double target_zmin,
    double target_xdd,     double target_ydd,     double target_zdd,
    int target_x_dim_glob, int target_y_dim_glob, int target_z_dim_glob,
    int cluster_num_sources, int cluster_idx_start,
    double *source_x, double *source_y, double *source_z, double *source_q,
    struct RunParams *run_params, double *potential, int gpu_async_stream_id)
{
    double eta = run_params->kernel_params[1];

    compute<<<1,32>>>();
    hipDeviceSynchronize();

    return;
}
