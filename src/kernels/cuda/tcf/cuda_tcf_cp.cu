#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

#include "cuda_tcf_cp.h"


__global__ 
static void compute(double kap, int batch_num_sources, int batch_idx_start,
    int cluster_q_start, int cluster_pts_start, int interp_order_lim,
    double *source_x, double *source_y, double *source_z, double *source_q,
    double *cluster_x, double *cluster_y, double *cluster_z, double *cluster_q)
{
    // printf("Hello TCF_CP thread %d, block %d\n", threadIdx.x, blockIdx.x);
}


__host__
void K_CUDA_TCF_CP_Lagrange(
    int batch_num_sources, int batch_idx_start, 
    int cluster_q_start, int cluster_pts_start, int interp_order_lim,
    double *source_x, double *source_y, double *source_z, double *source_q,
    double *cluster_x, double *cluster_y, double *cluster_z, double *cluster_q,
    struct RunParams *run_params, int gpu_async_stream_id)
{
    double kap = run_params->kernel_params[0];
    //double eta = run_params->kernel_params[1];
    //double kap_eta_2 = kap * eta / 2.0;

    compute<<<1,1>>>(kap, batch_num_sources, batch_idx_start,
                      cluster_q_start, cluster_pts_start, interp_order_lim,
                      source_x,  source_y,  source_z,  source_q,
                      cluster_x, cluster_y, cluster_z, cluster_q);
    hipDeviceSynchronize();

    return;
}
