#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

#include "cuda_tcf_pp.h"


__global__ 
static void compute()
{
    printf("Hello TCF_PP thread %d, block %d\n", threadIdx.x, blockIdx.x);
}


__host__
void K_CUDA_TCF_PP(
    int target_x_low_ind,  int target_x_high_ind,
    int target_y_low_ind,  int target_y_high_ind,
    int target_z_low_ind,  int target_z_high_ind,
    double target_xmin,    double target_ymin,    double target_zmin,
    double target_xdd,     double target_ydd,     double target_zdd,
    int target_x_dim_glob, int target_y_dim_glob, int target_z_dim_glob,
    int cluster_num_sources, int cluster_idx_start,
    double *source_x, double *source_y, double *source_z, double *source_q,
    struct RunParams *run_params, double *potential, int gpu_async_stream_id)
{
    double kap = run_params->kernel_params[0];
    double eta = run_params->kernel_params[1];
    double kap_eta_2 = kap * eta / 2.0;

    compute<<<1,32>>>();
    hipDeviceSynchronize();

    return;
}
