#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <stdio.h>

#include "cuda_coulomb_cp.h"


__global__ 
static void compute(int batch_num_sources, int batch_idx_start,
    int cluster_q_start, int cluster_pts_start, int interp_order_lim,
    double *source_x, double *source_y, double *source_z, double *source_q,
    double *cluster_x, double *cluster_y, double *cluster_z, double *cluster_q)
{
    printf("Hello Coulomb_CP thread %d, block %d\n", threadIdx.x, blockIdx.x);
}


__host__
void K_CUDA_Coulomb_CP_Lagrange(
    int batch_num_sources, int batch_idx_start, 
    int cluster_q_start, int cluster_pts_start, int interp_order_lim,
    double *source_x, double *source_y, double *source_z, double *source_q,
    double *cluster_x, double *cluster_y, double *cluster_z, double *cluster_q,
    struct RunParams *run_params, int gpu_async_stream_id)
{
    compute<<<1,32>>>(batch_num_sources, batch_idx_start,
                      cluster_q_start, cluster_pts_start, interp_order_lim,
                      source_x,  source_y,  source_z,  source_q,
                      cluster_x, cluster_y, cluster_z, cluster_q);
    hipDeviceSynchronize();

    return;
}
